
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <fstream>
#include <iostream>
#include <cmath>
#include <vector>
#include <algorithm>
#include <string>
#include <queue>

using namespace std;

__device__
double getValue(double *pre, int i, int j, int N) {
	return ((i >= 0 && j >= 0) ? pre[j * N + i] : 0);
}

__global__
void kernel(int N, int M, int n, int m, double th1, double th2, double avg, int *data, int *query, double *result, double *pre) {
    int corner_x = blockIdx.x * 32 + threadIdx.x;
	int corner_y = blockIdx.y * 32 + threadIdx.y;

	double gray = 0, x1, x2, y1, y2, cos, sin;

	if(blockIdx.z == 0) {
		cos = 0.707; sin = -0.707;
		x1 = corner_x; x2 = corner_x + (n - 1) * cos - (m - 1) * sin;
		y1 = corner_y + (n - 1) * sin; y2 = corner_y + (m - 1) * cos;
	} else if(blockIdx.z == 1) {
		cos = 1; sin = 0;
		x1 = corner_x; x2 = corner_x + n - 1;
		y1 = corner_y; y2 = corner_y + m - 1;
	} else {
		cos = 0.707; sin = 0.707;
		x1 = corner_x - (m - 1) * sin; x2 = corner_x + (n - 1) * cos;
		y1 = corner_y; y2 = corner_y + (n - 1) * sin + (m - 1) * cos;
	}

	if(floor(x1) < 0 || ceil(x2) >= N || floor(y1) < 0 || ceil(y2) >= M) {
		if(corner_y * N * 3 + corner_x * 3 + blockIdx.z < N*M*3) result[corner_y * N * 3 + corner_x * 3 + blockIdx.z] = INT_MAX;
		return;
	}

	gray = getValue(pre, floor(x2), floor(y2), N) - getValue(pre, floor(x2), ceil(y1) - 1, N) - getValue(pre, ceil(x1) - 1, floor(y2), N) + getValue(pre, ceil(x1) - 1, ceil(y1) - 1, N);
	gray /= ((floor(x2) - ceil(x1) + 1) * (floor(y2) - ceil(y1) + 1));

	if(abs(gray - avg) > th2) {
		result[corner_y * N * 3 + corner_x * 3 + blockIdx.z] = INT_MAX;
		return;
	}

	double rotate_x, rotate_y, scaled_x, scaled_y, interpolated, ans = 0;
	int query_data, p1_x, p1_y, p2_x, p2_y, p3_x, p3_y, p4_x, p4_y, z00, z01, z10, z11;

	for(int j = 0; j < m; j++) {
		for(int i = 0; i < n; i++) {
			for(int k = 0; k < 3; k++) {
				rotate_x = corner_x + i*cos - j*sin;
				rotate_y = corner_y + i*sin + j*cos;

				p1_x = floor(rotate_x), p1_y = floor(rotate_y);
				p2_x = floor(rotate_x), p2_y = ceil(rotate_y);
				p3_x = ceil(rotate_x), p3_y = floor(rotate_y);
				p4_x = ceil(rotate_x), p4_y = ceil(rotate_y);

				scaled_x = rotate_x - p1_x;
				scaled_y = rotate_y - p1_y;

				z00 = data[p1_y * N * 3 + p1_x * 3 + k];
				z01 = data[p2_y * N * 3 + p2_x * 3 + k];
				z10 = data[p3_y * N * 3 + p3_x * 3 + k];
				z11 = data[p4_y * N * 3 + p4_x * 3 + k];

				query_data = query[j * n * 3 + i * 3 + k];

				interpolated = z00 * (1 - scaled_x) * (1 - scaled_y) + z10 * scaled_x * (1 - scaled_y) + z01 * (1 - scaled_x) * scaled_y + z11 * scaled_x * scaled_y;
				ans += (interpolated - query_data) * (interpolated - query_data);
				if(ans > th1 * th1 * (m*n*3)) {
					result[corner_y * N * 3 + corner_x * 3 + blockIdx.z] = INT_MAX;
					return;
				}
			}
		}
	}

	ans = sqrt(ans / (m*n*3));
	result[corner_y * N * 3 + corner_x * 3 + blockIdx.z] = ans;
}

int main(int argc, char *argv[]) {
	int angles[] = {-45, 0, 45};

    ifstream fs_data, fs_query;
    fs_data.open(argv[1], ios::in);
	fs_query.open(argv[2], ios::in);

	int N, M, n, m;
	fs_data >> M >> N; fs_query >> m >> n;

	double th1 = stof(argv[3]), th2 = stof(argv[4]);
	int output_n = stoi(argv[5]);
	
	int *data, *query, *d_data, *d_query;
	double *result, *d_result, *pre, *d_pre;
	data = (int *)malloc(N*M*3*sizeof(int));
	query = (int *)malloc(n*m*3*sizeof(int));
	result = (double *)malloc(N*M*3*sizeof(double));
	pre = (double *)malloc(N*M*sizeof(double));
	hipMalloc(&d_data, N*M*3*sizeof(int));
	hipMalloc(&d_query, n*m*3*sizeof(int));
	hipMalloc(&d_result, N*M*3*sizeof(double));
	hipMalloc(&d_pre, N*M*sizeof(double));

	for(int j = M - 1; j >= 0; j--) {
		for(int i = 0; i < N; i++) {
			for(int k = 0; k < 3; k++) {
				fs_data >> data[j*N*3 + i*3 + k];
			}
		}
	}

	for(int j = m - 1; j >= 0 ; j--) {
		for(int i = 0; i < n; i++) {
			for(int k = 0; k < 3 ; k++) {
				fs_query >> query[j*n*3 + i*3 + k];
			}
		}
	}

	hipMemcpy(d_data, data, N*M*3*sizeof(int), hipMemcpyDefault);
	hipMemcpy(d_query, query, n*m*3*sizeof(int), hipMemcpyDefault);

	double avg = 0;
	for(int j = 0; j < m; j++) {
		for(int i = 0; i < n; i++) {
			avg += (double) (query[j * n * 3 + i * 3] + query[j * n * 3 + i * 3 + 1] + query[j * n * 3 + i * 3 + 2]) / 3;
		}
	}
	avg /= (n * m);

	for(int j = 0; j < M; j++) {
		for(int i = 0; i < N; i++) {
			if(i == 0 && j == 0) {
				pre[j * N + i] = (double) (data[j * N * 3 + i * 3] + data[j * N * 3 + i * 3 + 1] + data[j * N * 3 + i * 3 + 2]) / 3;
			} else if(i == 0) {
				pre[j * N + i] = pre[(j - 1) * N + i] + (double) (data[j * N * 3 + i * 3] + data[j * N * 3 + i * 3 + 1] + data[j * N * 3 + i * 3 + 2]) / 3;
			} else if(j == 0) {
				pre[j * N + i] = pre[j * N + (i - 1)] + (double) (data[j * N * 3 + i * 3] + data[j * N * 3 + i * 3 + 1] + data[j * N * 3 + i * 3 + 2]) / 3;
			} else {
				pre[j * N + i] = pre[(j - 1) * N + i] + pre[j * N + (i - 1)] - pre[(j - 1) * N + (i - 1)] + (double) (data[j * N * 3 + i * 3] + data[j * N * 3 + i * 3 + 1] + data[j * N * 3 + i * 3 + 2]) / 3;
			}
		}
	}

	hipMemcpy(d_pre, pre, N*M*sizeof(double), hipMemcpyDefault);

	dim3 num_blocks((N + 31) / 32, (M + 31) / 32, 3);
	dim3 num_threads(32, 32);

	kernel<<<num_blocks, num_threads>>>(N, M, n, m, th1, th2, avg, d_data, d_query, d_result, d_pre);

	hipMemcpy(result, d_result, N*M*3*sizeof(double), hipMemcpyDefault);

	if(output_n > 1) {
		vector<tuple<double,int, int, int>> v;
		for(int j=0; j< M ; j++){
			for(int i=0; i< N; i++){
				for(int k=0; k<3 ; k++){
					tuple<double, int, int, int> n1(result[j*N*3 + i*3 + k], i, j, k);
					v.push_back(n1);
				}
			}
		}
		priority_queue<tuple<double, int, int, int>, vector<tuple<double, int, int, int>>, greater<tuple<double, int, int, int>>> pq(v.begin(), v.end()); 
		for(int i = 0; i < output_n; i++) {
			tuple<double, int, int, int> t = pq.top();
			pq.pop();
			if( get<0>(t) > th1){
				break;
			}
			cout << get<2>(t) << " " << get<1>(t) << " " << angles[get<3>(t)] << "\n";
			// cout << get<0>(t) << "\n";
		}
		free(data); free(query); free(result); hipFree(d_data); hipFree(d_query); hipFree(d_result);
		return 0;
	}

	double min_val = (double) INT_MAX, I = -1, J = -1, K = -1;
	for(int j = 0; j < M; j++) {
		for(int i = 0; i < N; i++) {
			for(int k = 0; k < 3; k++) {
				if(result[j*N*3 + i*3 + k] < min_val) {
					min_val = result[j*N*3 + i*3 + k];
					I = i; J = j; K = k;
				}
			}
		}
	}

	free(data); free(query); free(result); hipFree(d_data); hipFree(d_query); hipFree(d_result);

	if(min_val>th1){
		return 0;
	}
	cout << J << " " << I << " " << angles[static_cast<int>(K)] << "\n";
	// cout << min_val << "\n";
}